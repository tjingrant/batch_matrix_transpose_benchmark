#include "hip/hip_runtime.h"

#include <algorithm>
#include <array>
#include <cassert>
#include <iostream>
#include <iomanip>

#include "util.h"

#define EIGEN_DEVICE_FUNC __host__ __device__
#define EIGEN_STRONG_INLINE __forceinline__

// TODO(mjanusz): Move this to a shared util file.
// A simple array that contains data that can be passed between CPU and GPU.
template <typename T, int IndexCount, T DefaultValue>
struct Array {
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE const T& operator[](int index) const {
    return data[index];
  }
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE T& operator[](int index) {
    return data[index];
  }
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Array() {
    for (int i = 0; i < IndexCount; i++) {
      data[i] = DefaultValue;
    }
  }
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Array(T a0) {
    data[0] = a0;
    for (int i = 1; i < IndexCount; i++) {
      data[i] = DefaultValue;
    }
  }
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Array(T a0, T a1) {
    data[0] = a0;
    data[1] = a1;
    for (int i = 2; i < IndexCount; i++) {
      data[i] = DefaultValue;
    }
  }
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Array(T a0, T a1, T a2) {
    data[0] = a0;
    data[1] = a1;
    data[2] = a2;
    for (int i = 3; i < IndexCount; i++) {
      data[i] = DefaultValue;
    }
  }
  EIGEN_STRONG_INLINE Array(const std::array<T, IndexCount>& array) {
    for (int i = 0; i < IndexCount; i++) {
      data[i] = array[i];
    }
  }
  T data[IndexCount];
};

// A dimension type with compile-time known size.
template <int IndexCount>
struct Dimension : Array<int, IndexCount, 1> {
  typedef Array<int, IndexCount, 1> Base;
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Dimension() : Base() {}
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Dimension(int a0) : Base(a0) {}
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Dimension(int a0, int a1)
      : Base(a0, a1) {}
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Dimension(int a0, int a1, int a2)
      : Base(a0, a1, a2) {}
  EIGEN_STRONG_INLINE Dimension(const std::array<int, IndexCount>& array)
      : Base(array) {}
};

// An index type with compile-time known size.
template <int IndexCount>
struct Index : Array<int, IndexCount, 0> {
  typedef Array<int, IndexCount, 0> Base;
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Index() : Base() {}
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Index(int a0) : Base(a0) {}
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Index(int a0, int a1) : Base(a0, a1) {}
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Index(int a0, int a1, int a2)
      : Base(a0, a1, a2) {}
};

// A helper function that converts a tensor index into a flat array index.
template <int IndexCount>
EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE int TensorIndexToFlat(
    const Index<IndexCount>& index, const Dimension<IndexCount>& dims) {
  int flat_index = index[0];
  for (int i = 1; i < IndexCount; i++) {
    flat_index = flat_index * dims[i] + index[i];
  }
  return flat_index;
}

// A helper function that converts a flat array index into a tensor index.
template <int IndexCount>
EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Index<IndexCount> FlatToTensorIndex(
    int index, const Dimension<IndexCount>& dims) {
  Index<IndexCount> tensor_index;
  for (int i = IndexCount - 1; i >= 0; i--) {
    tensor_index[i] = index % dims[i];
    index /= dims[i];
  }
  return tensor_index;
}

// A Cuda custom kernel that swaps dimension-0 and dimension-2 of a 3D tensor.
template <typename T>
__global__ void SwapDimension0And2InTensor3Simple(int nthreads, const T* input,
                                                  Dimension<3> input_dims,
                                                  T* output) {
  Dimension<3> output_dims;
  output_dims[0] = input_dims[2];
  output_dims[1] = input_dims[1];
  output_dims[2] = input_dims[0];

  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int output_index = index;

    Index<3> output_tensor_index = FlatToTensorIndex(output_index, output_dims);

    Index<3> input_tensor_index;
    input_tensor_index[0] = output_tensor_index[2];
    input_tensor_index[1] = output_tensor_index[1];
    input_tensor_index[2] = output_tensor_index[0];

    int input_index = TensorIndexToFlat(input_tensor_index, input_dims);

    output[output_index] = __ldg(input + input_index);
  }
}

// A Cuda custom kernel that swaps dimension-1 and dimension-2 of a 3D tensor.
template <typename T>
__global__ void SwapDimension1And2InTensor3Simple(int nthreads, const T* input,
                                                  Dimension<3> input_dims,
                                                  T* output) {
  Dimension<3> output_dims;
  output_dims[0] = input_dims[0];
  output_dims[1] = input_dims[2];
  output_dims[2] = input_dims[1];

  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int output_index = index;
    Index<3> output_tensor_index = FlatToTensorIndex(output_index, output_dims);

    Index<3> input_tensor_index;
    input_tensor_index[0] = output_tensor_index[0];
    input_tensor_index[1] = output_tensor_index[2];
    input_tensor_index[2] = output_tensor_index[1];

    int input_index = TensorIndexToFlat(input_tensor_index, input_dims);

    output[output_index] = __ldg(input + input_index);
  }
}

// Use shared memory tiles to swap dimension-1 and dimension-2 of a 3D tensor,
// where dimensions are zero-based: output[i][j][k] = input[i][k][j].
//
// Each thread block operates on a single tile, a square of dimensions TileSize
// x TileSize.  We require that the thread block's X dimension equals TileSize,
// and its Y dimension equals NumSubTiles.
//
// For best performance, you should probably set TileSize equal to the number of
// threads in a warp (32 in nvidia GPUs).  With a TileSize of 32, NumSubTiles ==
// 4 or 8 seems to get the best performance on K40 GPUs.
template <typename T, int TileSize, int NumSubTiles>
__global__ void SwapDimension1And2InTensor3UsingTiles(const T* input,
                                                      Dimension<3> input_dims,
                                                      T* output) {
  // One extra line in the inner dimension to avoid share memory bank conflict.
  __shared__ T shared_memory_tile[TileSize][TileSize + 1];

  static_assert(TileSize % NumSubTiles == 0,
                "TileSize must be divisible by NumSubTiles");


//#define MY_ASSERT(X) if (!(X)) printf(#X)
#define MY_ASSERT(X)
  MY_ASSERT(blockDim.x == TileSize);
  MY_ASSERT(blockDim.y == NumSubTiles);
  MY_ASSERT(blockDim.z == 1);
  MY_ASSERT(gridDim.y == 1);
  MY_ASSERT(gridDim.z == 1);


  // We break down the tile into NumSubTiles groups, so each thread processes
  // kSubTileSize elements (except at the edges of the input).
  const int kSubTileSize = TileSize / NumSubTiles;

  int x = threadIdx.x;

  Dimension<3> output_dims = {
      input_dims[0], input_dims[2], input_dims[1],
  };

  Dimension<3> input_dims_in_tiles = {
      input_dims[0], (input_dims[1] + TileSize - 1) / TileSize,
      (input_dims[2] + TileSize - 1) / TileSize,
  };

  Index<3> input_tile_index =
      FlatToTensorIndex(blockIdx.x, input_dims_in_tiles);

  Index<3> input_tile_origin = {
      input_tile_index[0], input_tile_index[1] * TileSize,
      input_tile_index[2] * TileSize,
  };

  int input_origin_flat_index =
      TensorIndexToFlat(input_tile_origin, input_dims);

  int tile_width = TileSize;
  // Only the last row or column may not have the full size.
  if (input_tile_index[2] == input_dims_in_tiles[2] - 1) {
    tile_width = input_dims[2] - (input_dims_in_tiles[2] - 1) * TileSize;
  }
  int tile_height = TileSize;
  if (input_tile_index[1] == input_dims_in_tiles[1] - 1) {
    tile_height = input_dims[1] - (input_dims_in_tiles[1] - 1) * TileSize;
  }

  int input_flat_index = input_origin_flat_index + x;
  int y_start = static_cast<int>(threadIdx.y) * kSubTileSize;

  // Load the data from input memory to the shared memory tile.
  if (x < tile_width) {
    int y_end = min(y_start + kSubTileSize, tile_height);
    for (int y = y_start; y < y_end; y++) {
      shared_memory_tile[y][x] = input[input_flat_index + y * input_dims[2]];
    }
  }

  __syncthreads();

  Index<3> output_tile_index = {
      input_tile_index[0], input_tile_index[2], input_tile_index[1],
  };

  Index<3> output_tile_origin = {
      output_tile_index[0], output_tile_index[1] * TileSize,
      output_tile_index[2] * TileSize,
  };

  int output_origin_flat_index =
      TensorIndexToFlat(output_tile_origin, output_dims);

  int output_flat_index = output_origin_flat_index + x;

  // Load the data from the shared memory tile to the output memory.
  if (x < tile_height) {
    int y_end = min(y_start + kSubTileSize, tile_width);
    for (int y = y_start; y < y_end; y++) {
      output[output_flat_index + y * output_dims[2]] = shared_memory_tile[x][y];
    }
  }
}

template <typename T, int THREAD_NUM>
__global__ void MySwapDimension1And2InTensor3UsingTiles(const T* __restrict__ input,
                                        Dimension<3> input_dims,
                                        T* __restrict__ output,
                                        int TILE_SIZE_I,
                                        int TILE_SIZE_J) {

  const int READ_ROW_PER_PASS = (THREAD_NUM/TILE_SIZE_J);
  const int WRITE_ROW_PER_PASS = (THREAD_NUM/TILE_SIZE_I);
  // One extra line in the inner dimension to avoid share memory bank conflict.
  __shared__ T shared_memory_tile[];

  #define SHARED(i, j)\
    shared_memory_tile[i*(TILE_SIZE_J+1)+j]

  int x = threadIdx.x;

  Dimension<3> output_dims = {
      input_dims[0], input_dims[2], input_dims[1],
  };

  Dimension<3> input_dims_in_tiles = {
      input_dims[0], (input_dims[1] + (TILE_SIZE_I) - 1) / (TILE_SIZE_I),
      (input_dims[2] + (TILE_SIZE_J) - 1) / (TILE_SIZE_J),
  };

  Index<3> input_tile_index =
      FlatToTensorIndex(blockIdx.x, input_dims_in_tiles);

  Index<3> input_tile_origin = {
      input_tile_index[0], input_tile_index[1] * (TILE_SIZE_I),
      input_tile_index[2] * (TILE_SIZE_J),
  };

  int input_origin_flat_index =
      TensorIndexToFlat(input_tile_origin, input_dims);

  #define INPUT(i, j)\
      input[input_origin_flat_index +\
        (i) * input_dims[2] + (j)]

  int tile_width = TILE_SIZE_J;
  // Only the last row or column may not have the full size.
  if (input_tile_index[2] == input_dims_in_tiles[2] - 1) {
    tile_width = input_dims[2] - (input_dims_in_tiles[2] - 1) * TILE_SIZE_J;
  }
  int tile_height = TILE_SIZE_I;
  if (input_tile_index[1] == input_dims_in_tiles[1] - 1) {
    tile_height = input_dims[1] - (input_dims_in_tiles[1] - 1) * TILE_SIZE_I;
  }

  int effective_thread_num = THREAD_NUM / TILE_SIZE_J * TILE_SIZE_J;

  if (x < effective_thread_num) {
    int ti = x/TILE_SIZE_J;
    int tj = x%TILE_SIZE_J;
    if (tj < tile_width)
      for (int i_loc = ti; i_loc < (tile_height); i_loc += READ_ROW_PER_PASS) {
        SHARED(i_loc, tj) = INPUT(i_loc, tj);
      }
  }

  __syncthreads();

  Index<3> output_tile_index = {
      input_tile_index[0], input_tile_index[2], input_tile_index[1],
  };

  Index<3> output_tile_origin = {
      output_tile_index[0], output_tile_index[1] * TILE_SIZE_J,
      output_tile_index[2] * TILE_SIZE_I,
  };

  int output_origin_flat_index =
      TensorIndexToFlat(output_tile_origin, output_dims);

  // Oriented with respect to the output array.
  effective_thread_num = THREAD_NUM / TILE_SIZE_I * TILE_SIZE_I;

  if (x < effective_thread_num) {
    int ti = x/TILE_SIZE_I;
    int tj = x%TILE_SIZE_I;

    #define OUTPUT(i, j)\
        output[output_origin_flat_index +\
          (i) * output_dims[2] + (j)]
    if (tj < tile_height)
      for (int i_loc = ti; i_loc < (tile_width); i_loc += WRITE_ROW_PER_PASS) {
        OUTPUT(i_loc, tj) = SHARED(tj, i_loc);
      }
  }
}

// Launch the GPU kernel that would swap dimension-1 and dimension-2 in a
// 3D tensor. It looks at the shape of the incoming data, and decides the best
// strategy to launch.
template <typename T>
void RunSwapDimension1And2InTensor3(const T* input,
                                    const Dimension<3>& input_dims, T* output) {
  // If both dimensions are not trivial, use tiles for the actual swapping.
  // Otherwise, the trivial swapping relying on the ldg cache is more efficient.
  static const int kMinDimensionToUseTiles = 16;
  static const int TileSize = 32;
  static const int NumSubTiles = 8;
  bool use_tiles = (input_dims[1] >= kMinDimensionToUseTiles &&
                    input_dims[2] >= kMinDimensionToUseTiles);

  Dimension<3> input_dims_in_tiles = {
        input_dims[0], (input_dims[1] + TileSize - 1) / TileSize,
        (input_dims[2] + TileSize - 1) / TileSize,
  };

  int total_tiles_count = input_dims_in_tiles[0] * input_dims_in_tiles[1] *
                            input_dims_in_tiles[2];
  if (use_tiles) {
    // We get best performance when TileSize is the number of threads in a warp
    // (32 on our GPUs) and NumSubTiles is 8, so our block size is 8 * 32 = 256
    // threads.
    dim3 griddim(total_tiles_count, 1, 1);
    dim3 blockdim(TileSize, NumSubTiles, 1);
    SwapDimension1And2InTensor3UsingTiles<T, TileSize, NumSubTiles><<<griddim, blockdim>>>(input, input_dims, output);
  } else {
    int total_element_count = input_dims[0] * input_dims[1] * input_dims[2];
    CudaLaunchConfig config = GetCudaLaunchConfig(total_element_count);
    SwapDimension1And2InTensor3Simple<T>
        <<<config.block_count, config.thread_per_block>>>(
            config.virtual_thread_count, input, input_dims, output);
  }
}

// Launch the GPU kernel that would swap dimension-1 and dimension-2 in a
// 3D tensor. It looks at the shape of the incoming data, and decides the best
// strategy to launch.
template <typename T>
void MyRunSwapDimension1And2InTensor3(const T* input,
                                    const Dimension<3>& input_dims, T* my_output) {
  // If both dimensions are not trivial, use tiles for the actual swapping.
  // Otherwise, the trivial swapping relying on the ldg cache is more efficient.
  static const int kMinDimensionToUseTiles = 16;
  static const int TileSize = 32;
  static const int NumSubTiles = 8;
  bool use_tiles = (input_dims[1] >= kMinDimensionToUseTiles &&
                    input_dims[2] >= kMinDimensionToUseTiles);

  if (use_tiles) {
    Dimension<3> my_input_dims_in_tiles = {
          input_dims[0], (input_dims[1] + TileSize - 1) / TileSize,
          (input_dims[2] + TileSize - 1) / TileSize,
    };

    int my_total_tiles_count = my_input_dims_in_tiles[0] * my_input_dims_in_tiles[1] *
                              my_input_dims_in_tiles[2];
    int THREAD_NUM = 32;
    MySwapDimension1And2InTensor3UsingTiles<float, 256, 32, 32><<<
          my_total_tiles_count, THREAD_NUM>>>(input, input_dims, my_output);

  } else {

    int tile_size_i = input_dims[1] >= kMinDimensionToUseTiles ? 128 : input_dims[1];
    int tile_size_j = input_dims[1] >= kMinDimensionToUseTiles ? input_dims[2] : 128;

    int THREAD_NUM = 128;
    Dimension<3> my_input_dims_in_tiles = {
          input_dims[0], (input_dims[1] + tile_size_i - 1) / tile_size_i,
          (input_dims[2] + tile_size_j - 1) / tile_size_j,
    };
    int my_total_tiles_count = my_input_dims_in_tiles[0] * my_input_dims_in_tiles[1] *
                              my_input_dims_in_tiles[2];

    #define LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(TILE_SIZE_I, TILE_SIZE_J) \
      if (tile_size_i == TILE_SIZE_I && tile_size_j == TILE_SIZE_J) \
        MySwapDimension1And2InTensor3UsingTiles<float, 128, TILE_SIZE_I, TILE_SIZE_J><<< \
              my_total_tiles_count, THREAD_NUM>>>(input, input_dims, my_output);

    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(128,   1)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(1  , 128)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(128,   2)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(2  , 128)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(128,   3)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(3  , 128)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(128,   4)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(4  , 128)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(128,   5)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(5  , 128)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(128,   6)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(6  , 128)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(128,   7)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(7  , 128)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(128,   8)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(8  , 128)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(128,   9)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(9  , 128)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(128,  10)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(10 , 128)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(128,  11)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(11 , 128)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(128,  12)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(12 , 128)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(128,  13)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(13 , 128)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(128,  14)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(14 , 128)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(128,  15)
    LAUNCH_MY_SWAP_DIMENSION_1_AND_2_IN_TENSOR_3_USING_TILES(15 , 128)

  }
}

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 } else { }\
}

int test(int N, int M, int P)
{
  printf("TEST PARAM %d, %d, %d\n", N, M, P);
  float *input_host, *output_host, *my_output_host;
  int size = N*M*P*sizeof(float);

  input_host = (float*)malloc(size);
  output_host = (float*)malloc(size);
  my_output_host = (float*)malloc(size);
  for (int i=0; i<N*M*P; i++) {
    input_host[i] = (float)i;
  }

  Dimension<3> input_dims = {
      N, M, P
  };

  float *input_device, *output_device, *my_output_device;
  hipMalloc((void**)&input_device, size);
  hipMalloc((void**)&output_device, size);
  hipMalloc((void**)&my_output_device, size);
  hipMemcpy(input_device, input_host, size, hipMemcpyHostToDevice );

#define BENCHMARK(X, REPEAT, NAME) \
  do {\
  float time; \
  hipEvent_t start, stop; \
  hipEventCreate(&start); \
  hipEventCreate(&stop); \
  hipEventRecord(start, 0); \
  for (int repeat=0; repeat<REPEAT; repeat++)\
  X;\
  hipEventRecord(stop, 0);\
  hipEventSynchronize(stop);\
  hipEventElapsedTime(&time, start, stop);\
  printf(NAME":  %f ms \t", time/(float)REPEAT);} while(0)\

  BENCHMARK(MyRunSwapDimension1And2InTensor3(input_device, input_dims, my_output_device), 100, "UNIFIED");
  BENCHMARK(RunSwapDimension1And2InTensor3(input_device, input_dims, output_device), 100, "SEPARATE");

  printf("\n");
  hipMemcpy(output_host, output_device, size, hipMemcpyDeviceToHost);
  hipMemcpy(my_output_host, my_output_device, size, hipMemcpyDeviceToHost);
  cudaCheckError();

  bool check = 1;
  for (int i=0; i<N; i++)
    for (int j=0; j<M; j++)
      for (int k=0; k<P; k++)
        check &= (output_host[i*M*P+j*P+k] == my_output_host[i*M*P+j*P+k]);

  assert(check);
  return check;
}

int main() {
  for (int i=2; i<16; i++) {
    test(128, i, 784);
    test(128, 784, i);
  }
  return 0;
}